#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <errno.h>

#include "hip/hip_runtime.h"


long POLYGON_DIM; //размер полигона = POLYGON_DIM * POLYGON_DIM

__global__ void markup(int* data, int* res, long* size) {
    int h = 3;
    if(gridDim.x * blockDim.x < (*size) * (*size)) { //если общее количество нитей меньше количества элементов полигона
        int tid = threadIdx.x + blockIdx.x * blockDim.x; //номер нити
        int number = ((*size) * (*size)) / (gridDim.x * blockDim.x); //количество элементов полигона, просматриваемых данной нитью
        int start_idx = tid * number; //начальный индекс элемента
        int finish_idx = start_idx + number; //конечный индекс элемента
        while (start_idx < finish_idx) {
            int curr = data[start_idx];
            if( start_idx / (*size) != 0     && curr - data[start_idx - (*size)] > h ||
                start_idx / (*size) != (*size) - 1 && curr - data[start_idx + (*size)] > h ||
                start_idx % (*size) != 0     && curr - data[start_idx - 1] > h ||
                start_idx % (*size) != (*size) - 1 && curr - data[start_idx + 1] > h ) 
                    res[start_idx] = -1;
            start_idx++;
        }
    } else { //если общее количество нитей больше количества элементов полигона
        int tid = threadIdx.x + blockIdx.x * blockDim.x; //номер нити
        if(tid < (*size) * (*size)) { //обязательно проверить, что не происходит выход за границы полигона
            int curr = data[tid];
            if( tid / (*size) != 0     && curr - data[tid - (*size)] > h ||
                tid / (*size) != (*size) - 1 && curr - data[tid + (*size)] > h ||
                tid % (*size) != 0     && curr - data[tid - 1] > h ||
                tid % (*size) != (*size) - 1 && curr - data[tid + 1] > h ) 
                    res[tid] = -1;
        }
    }
}

__host__ void fill(int* P) {
    for(int i = 0; i < POLYGON_DIM * POLYGON_DIM; i++) {
        P[i] = rand() % 10 + 1;
    }
}

//первый параметр - размер полигона
//второй параметр - количество блоков
//третий параметр - количество нитей
int main(int argc, const char* argv[]) {

    if (argc != 4) {
		fprintf(stderr, "Incorrect number of arguments\n");
		return -1;
	}

    POLYGON_DIM = strtol(argv[1], NULL, 10);
    long BLOCK_NUMBER = strtol(argv[2], NULL, 10); //количество блоков
    long THREAD_NUMBER = strtol(argv[3], NULL, 10); //количество нитей в блоке

    int* P;
    int* original_P;
    int* res_P;
    long* size;

    P = (int*) malloc( POLYGON_DIM * POLYGON_DIM * sizeof(int) );
    hipMalloc( (void**)&original_P, POLYGON_DIM * POLYGON_DIM * sizeof(int) );
    hipMalloc( (void**)&res_P, POLYGON_DIM * POLYGON_DIM * sizeof(int) );
    hipMalloc( (void**)&size, sizeof(long) );

    srand( time( 0 ) );
    fill(P);
    hipMemcpy( original_P, P, POLYGON_DIM * POLYGON_DIM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( res_P, original_P, POLYGON_DIM * POLYGON_DIM * sizeof(int), hipMemcpyDeviceToDevice);
    hipMemcpy( size, &POLYGON_DIM, sizeof(long), hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    markup<<< BLOCK_NUMBER, THREAD_NUMBER >>>(original_P, res_P, size);

    hipEventSynchronize(stop);
    hipEventRecord(stop, 0);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed Time: %3.1f ms\n", elapsedTime);

    for(int i = 0; i < POLYGON_DIM; i++) {
        for(int j = 0; j < POLYGON_DIM; j++) {
            printf("%d ", P[POLYGON_DIM*i + j]);
        }
        printf("\n");
    }
    printf("\n");

    hipMemcpy( P, res_P, POLYGON_DIM * POLYGON_DIM * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < POLYGON_DIM; i++) {
        for(int j = 0; j < POLYGON_DIM; j++) {
            printf("%d ", P[POLYGON_DIM*i + j]);
        }
        printf("\n");
    }

    free(P);
    hipFree(original_P);
    hipFree(res_P);
    hipFree(size);

    return 0;
}